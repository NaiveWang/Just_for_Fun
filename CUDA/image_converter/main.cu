#include "hip/hip_runtime.h"
#include <stdio.h>
//#include "bmp_loader.h"
//#include "converter.h"
//#include <unistd.h>
#include "bmp_lib.h"
#include <sys/time.h>
#include <stdlib.h>
#define LINEAR_R 0.2126
#define LINEAR_G 0.7152
#define LINEAR_B 0.0722

bmpa* loadimage_rgb(char* file_name)
{
  FILE* fp=NULL;
  bmpa* a = (bmpa*)malloc(sizeof(bmpa));

  //open file
  fp=fopen(file_name,"rb");
  //read header
  fread(a->header,sizeof(unsigned char),54,fp);
  //get height and width
  a->w=*(int*)(a->header+18);
  a->h=*(int*)(a->header+22);
  //allocate
  a->type = BMP_RGB;
  a->array = malloc(sizeof(unsigned char)*3*a->w*a->h);
  //read content
  fread(a->array,sizeof(unsigned char),3*a->w*a->h,fp);
  //close file
  fclose(fp);
  //return pointer
  return a;
}
void free_image(bmpa* a)
{
  free(a->array);
  free(a);
}
void saveimage_grey(bmpa* a,char* file_name)
{
  FILE *fp=NULL;
  fp = fopen(file_name,"wb");
  fwrite(a->header,sizeof(char),54,fp);
  fwrite(a->array,sizeof(char)*3,a->w*a->h,fp);
  //fwrite((int*)EOF,sizeof(int),1,fp);
  fclose(fp);
}

__global__ void gpu_thread_greyscaler(unsigned char** base)
{
  int counter=8000*8000;
  while(counter--)
  {
    *(unsigned char*)(base + 3 * counter) =
    *(unsigned char*)(base + 3 * counter + 1) =
    *(unsigned char*)(base + 3 * counter + 2) =
    (unsigned char)(
      (*(unsigned char*)(base + 3 * counter)) * LINEAR_R +
      (*(unsigned char*)(base + 3 * counter + 1)) * LINEAR_G +
      (*(unsigned char*)(base + 3 * counter + 2)) * LINEAR_B);
  }
  /*
  int i=threadIdx.x;
  *(unsigned char*)(base+i*3) =
  *(unsigned char*)(base+i*3 + 1) =
  *(unsigned char*)(base+i*3 + 2) =
  (unsigned char)(
    (*(unsigned char*)(base+i*3)) * LINEAR_R +
    (*(unsigned char*)(base+i*3 + 1)) * LINEAR_G +
    (*(unsigned char*)(base+i*3 + 2)) * LINEAR_B);*/
}

int convert_gpu(bmpa* a)
{
  unsigned char* dev=NULL;
  long stat;
  //printf("N : %d",sizeof(char)*a->w*a->h*3);
  stat = hipMalloc(&dev,sizeof(char)*a->w*a->h*3);
  if(stat==hipSuccess)
  {
    stat = hipMemcpy(dev,a->array,sizeof(char)*a->w*a->h*3,hipMemcpyHostToDevice);
    printf("W : %ld\n",stat);
    //computing section
    gpu_thread_greyscaler<<<1,1>>>(&dev);
    stat = hipMemcpy(a->array,dev,sizeof(char)*a->w*a->h*3,hipMemcpyDeviceToHost);
    //printf("W : %ld\n",stat);
    hipFree(dev);
    return 0;
  }
  else
  {
    printf("E : %ld\n",stat);
    return 1;
  }

}
double get_time()
{
  static struct timeval t;

  gettimeofday(&t,NULL);
  return t.tv_sec*1000.0 + t.tv_usec/1000.0;
}

void convert_cpu(bmpa* a)
{
  int counter = a->w * a->h;
  while(counter--)
  {
    *(unsigned char*)(a->array + 3 * counter) =
    *(unsigned char*)(a->array + 3 * counter + 1) =
    *(unsigned char*)(a->array + 3 * counter + 2) =
    (unsigned char)(
      (*(unsigned char*)(a->array + 3 * counter)) * LINEAR_R +
      (*(unsigned char*)(a->array + 3 * counter + 1)) * LINEAR_G +
      (*(unsigned char*)(a->array + 3 * counter + 2)) * LINEAR_B);
  }
}
int main(int argc, char *argv[])
{
  bmpa *bp=NULL;
  double t;
  switch(argv[1][0])
  {
    case 'g':
      printf("Loading file...");
      t=get_time();
      bp=loadimage_rgb("image.bmp");
      t=get_time()-t;
      printf("Time Elapsed:%lfms\n",t);

      printf("Computing using GPU...");
      t=get_time();
      convert_gpu(bp);
      t=get_time()-t;
      printf("Time Elapsed:%lfms\n",t);

      //printf("<>%d<>%d<>\n",bp->w,bp->h);

      printf("Saving file...");
      t=get_time();
      saveimage_grey(bp,"grey.bmp");
      t=get_time()-t;
      printf("Time Elapsed:%lfms\n",t);
      free_image(bp);
      return 0;
    case 'c':
      printf("Loading file...");
      t=get_time();
      bp=loadimage_rgb("image.bmp");
      t=get_time()-t;
      printf("Time Elapsed:%lfms\n",t);

      printf("Computing using CPU...");
      t=get_time();
      convert_cpu(bp);
      t=get_time()-t;
      printf("Time Elapsed:%lfms\n",t);

      //printf("<>%d<>%d<>\n",bp->w,bp->h);

      printf("Saving file...");
      t=get_time();
      saveimage_grey(bp,"grey.bmp");
      t=get_time()-t;
      printf("Time Elapsed:%lfms\n",t);
      free_image(bp);
      return 0;
    default:
      return -1;
  }
  //return 0;
}
