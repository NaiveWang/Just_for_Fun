#include "hip/hip_runtime.h"
#include <stdio.h>
//#include "bmp_loader.h"
//#include "converter.h"
//#include <unistd.h>
#include "bmp_lib.h"
#include <sys/time.h>
#include <stdlib.h>
#define LINEAR_R 0.2126f
#define LINEAR_G 0.7152f
#define LINEAR_B 0.0722f
#define THREADS 1024

double get_time()
{
  static struct timeval t;

  gettimeofday(&t,NULL);
  return t.tv_sec*1000.0 + t.tv_usec/1000.0;
}


bmpa* loadimage_rgb(char* file_name)
{
  FILE* fp=NULL;
  bmpa* a = (bmpa*)malloc(sizeof(bmpa));

  //open file
  fp=fopen(file_name,"rb");
  //read header
  fread(a->header,sizeof(unsigned char),54,fp);
  //get height and width
  a->w=*(int*)(a->header+18);
  a->h=*(int*)(a->header+22);
  //allocate
  a->type = BMP_RGB;
  a->array = malloc(sizeof(unsigned char)*3*a->w*a->h);
  //read content
  fread(a->array,sizeof(unsigned char),3*a->w*a->h,fp);
  //close file
  fclose(fp);
  //return pointer
  return a;
}
void free_image(bmpa* a)
{
  free(a->array);
  free(a);
}
void saveimage_grey(bmpa* a,char* file_name)
{
  FILE *fp=NULL;
  fp = fopen(file_name,"wb");
  fwrite(a->header,sizeof(char),54,fp);
  fwrite(a->array,sizeof(char)*3,a->w*a->h,fp);
  //fwrite((int*)EOF,sizeof(int),1,fp);
  fclose(fp);
}

__global__ void gpu_thread_greyscaler(unsigned char* base,int N)
{

  int i=blockDim.x * blockIdx.x + threadIdx.x;
  if(i<N)
  {
    *(unsigned char*)(base+i*3) =
    *(unsigned char*)(base+i*3 + 1) =
    *(unsigned char*)(base+i*3 + 2) =
    (
      (*(unsigned char*)(base+i*3)) * LINEAR_R +
      (*(unsigned char*)(base+i*3 + 1)) * LINEAR_G +
      (*(unsigned char*)(base+i*3 + 2)) * LINEAR_B);
  }
}

int convert_gpu(bmpa* a)
{
  double t;
  unsigned char* dev=NULL;
  long stat;
  //printf("N : %d",sizeof(char)*a->w*a->h*3);
  t=get_time();
  stat = hipMalloc(&dev,sizeof(char)*a->w*a->h*3);
  t=get_time()-t;
  printf("Memory Allocation Delay:%lfms\n",t);
  if(stat==hipSuccess)
  {
    t=get_time();
    stat = hipMemcpy(dev,a->array,sizeof(char)*a->w*a->h*3,hipMemcpyHostToDevice);
    //printf("W : %ld\n",stat);
    //computing section
    gpu_thread_greyscaler<<<(a->w*a->h+THREADS-1)/THREADS,THREADS>>>(dev,a->w*a->h);
    //getchar();

    stat = hipMemcpy(a->array,dev,sizeof(char)*a->w*a->h*3,hipMemcpyDeviceToHost);
    t=get_time()-t;
    printf("Time Elapsed:%lfms\n",t);
    hipFree(dev);
    return 0;
  }
  else
  {
    printf("E : %ld\n",stat);
    return 1;
  }

}


void convert_cpu(bmpa* a)
{
  int counter = a->w * a->h;
  while(counter--)
  {
    *((unsigned char*)a->array + 3 * counter) =
    *((unsigned char*)a->array + 3 * counter + 1) =
    *((unsigned char*)a->array + 3 * counter + 2) =
    (unsigned char)(
      (*((unsigned char*)a->array + 3 * counter)) * LINEAR_R +
      (*((unsigned char*)a->array + 3 * counter + 1)) * LINEAR_G +
      (*((unsigned char*)a->array + 3 * counter + 2)) * LINEAR_B);
  }
}
int main(int argc, char *argv[])
{
  bmpa *bp=NULL;
  double t;
  switch(argv[1][0])
  {
    case 'c':
      printf("Loading file...");
      t=get_time();
      bp=loadimage_rgb(argv[2]);
      t=get_time()-t;
      printf("Time Elapsed:%lfms\n",t);

      printf("Computing using CPU\n");
      t=get_time();
      convert_cpu(bp);
      t=get_time()-t;
      printf("Time Elapsed:%lfms\n",t);

      //printf("<>%d<>%d<>\n",bp->w,bp->h);

      printf("Saving file...");
      t=get_time();
      saveimage_grey(bp,(char*)"grey.bmp");
      t=get_time()-t;
      printf("Time Elapsed:%lfms\n",t);
      free_image(bp);
      return 0;
    case 'g':
      printf("Loading file...");
      t=get_time();
      bp=loadimage_rgb(argv[2]);
      t=get_time()-t;
      printf("Time Elapsed:%lfms\n",t);

      printf("Computing using GPU\n");

      t=get_time();
      convert_gpu(bp);
      t=get_time()-t;
      printf("Total:%lfms\n",t);

      //printf("<>%d<>%d<>\n",bp->w,bp->h);

      printf("Saving file...");
      t=get_time();
      saveimage_grey(bp,(char*)"grey.bmp");
      t=get_time()-t;
      printf("Time Elapsed:%lfms\n",t);
      free_image(bp);
      return 0;
    default:
      return -1;
  }
  //return 0;
}
