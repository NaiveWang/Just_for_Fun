/* This C file will be compiled by nvcc */

#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void add(int a,int b,int *c)
{
  *c = a+b;
}
int main()
{
  int a,b,c;
  int *dev_c;

  a=10;
  b=100;

  hipMalloc(&dev_c,sizeof(int));

  add<<<1,1>>>(a,b,dev_c);

  hipMemcpy(&c,dev_c,sizeof(int),hipMemcpyDeviceToHost);

  printf(">>>%d\n",c);

  hipFree(dev_c);
  return 0;
}
