/* This C file will be compiled by nvcc */

#include <hip/hip_runtime.h>
#include "stdio.h"


unsigned char a[10]={2,3,4,5,6,7,8,9,10,1};
unsigned char b[10]={1,2,3,4,5,6,7,8,9,10};
__global__ void add(unsigned char* a,unsigned char* b,unsigned char* c,int n)
{
  int i=blockDim.x * blockIdx.x + threadIdx.x;
  if(i<n)
    c[i] = a[i]*0.7f + b[i]*0.3f;
}

int main()
{
  unsigned char* dev_c=NULL;
  unsigned char* dev_a=NULL;
  unsigned char* dev_b=NULL;
  unsigned char c[10];

  int counter;

  long status  = 0;
  hipMalloc(&dev_a,sizeof(unsigned char)*10);
  hipMalloc(&dev_b,sizeof(unsigned char)*10);
  status = hipMalloc(&dev_c,sizeof(unsigned char)*10);
  printf("%ld<<\n",status);
  if(status == hipSuccess)
  {
    hipMemcpy(dev_a,a,sizeof(unsigned char)*10,hipMemcpyHostToDevice);
    hipMemcpy(dev_b,b,sizeof(unsigned char)*10,hipMemcpyHostToDevice);

    add<<<10/256+1,256>>>(dev_a,dev_b,dev_c,10);

    hipMemcpy(c,dev_c,sizeof(unsigned char)*10,hipMemcpyDeviceToHost);

    for(counter=0;counter<10;counter++)
    {
      printf("%d ",c[counter]);
    }
    printf("\n");

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
  }
  else
  {
    printf("%ld>Allocating failed\n",(long)dev_c);
  }
  return 0;
}
