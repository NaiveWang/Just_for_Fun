/* This C file will be compiled by nvcc */

#include <hip/hip_runtime.h>
#include "stdio.h"


__global__ void add(float a,float b,float* c)
{
  *c = a + b;
}

int main()
{
  float a,b,c;
  float *dev_c=NULL;
  long status  = 0;

  a=1.2f;
  b=5.1f;

  status = hipMalloc(&dev_c,sizeof(float));
  printf("%ld<<\n",status);
  if(status == hipSuccess)
  {
    add<<<1,1>>>(a,b,dev_c);

    hipMemcpy(&c,dev_c,sizeof(float),hipMemcpyDeviceToHost);

    printf(">>>%f\n",c);

    hipFree(dev_c);
  }
  else
  {
    printf("%ld>Allocating failed\n",(void*)dev_c);
  }
  return 0;
}
