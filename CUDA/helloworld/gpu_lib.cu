#include "hip/hip_runtime.h"
#include "gpu_lib.cuh"
__global__ void add(unsigned char* a,unsigned char* b,unsigned char* c,int n)
{
  int i=blockDim.x * blockIdx.x + threadIdx.x;
  if(i<n)
    c[i] = a[i]*0.7f + b[i]*0.3f;
}
extern "C" void func(unsigned char* a,unsigned char *b,unsigned char *c)
{
  unsigned char* dev_c=NULL;
  unsigned char* dev_a=NULL;
  unsigned char* dev_b=NULL;

  long status  = 0;
  hipMalloc(&dev_a,sizeof(unsigned char)*10);
  hipMalloc(&dev_b,sizeof(unsigned char)*10);
  status = hipMalloc(&dev_c,sizeof(unsigned char)*10);
  //printf("%ld<<\n",status);
  if(status == hipSuccess)
  {
    hipMemcpy(dev_a,a,sizeof(unsigned char)*10,hipMemcpyHostToDevice);
    hipMemcpy(dev_b,b,sizeof(unsigned char)*10,hipMemcpyHostToDevice);

    add<<<10/256+1,256>>>(dev_a,dev_b,dev_c,10);

    hipMemcpy(c,dev_c,sizeof(unsigned char)*10,hipMemcpyDeviceToHost);



    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
  }
}
